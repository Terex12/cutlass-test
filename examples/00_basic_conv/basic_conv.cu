#include "hip/hip_runtime.h"

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>

// Helper methods to check for errors
#include "helper.h"

//
// CUTLASS includes needed for single-precision GEMM kernel
//

// Defines cutlass::conv::device::Conv, the generic Conv computation template class.
#include "cutlass/conv/device/conv.h"

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSconvNN(
        int NF, int NY, int NX, int NH, int NW, int NR, int NS, int NC,
        int sW, int sH,
        float alpha,
        float const *A,
        int lda0, int lda1, int lda2,
        float const *B,
        int ldb0, int ldb1, int ldb2,
        float beta,
        float *C,
        int ldc0, int ldc1, int ldc2) {

    // Define type definition for single-precision CUTLASS GEMM with column-major
    // input matrices and 128x128x8 threadblock tile size (chosen by default).
    //
    // To keep the interface manageable, several helpers are defined for plausible compositions
    // including the following example for single-precision GEMM. Typical values are used as
    // default template arguments. See `cutlass/conv/device/default_conv_configuration.h` for more details.
    //
    // To view the full conv device API interface, see `cutlass/conv/device/conv.h`

    using TensorNCHW = cutlass::layout::TensorNCHW;

    using CutlassConv = cutlass::conv::device::Conv<float,        // Data-type of A matrix
            TensorNCHW,  // Layout of A matrix
            float,        // Data-type of B matrix
            TensorNCHW,  // Layout of B matrix
            float,        // Data-type of C matrix
            TensorNCHW>; // Layout of C matrix

    /// Stride vector
    using Stride = Coord<3, Index>;

    /// Construct stride for tensors
    Stride strideA = make_Coord(lda0, lda1, lda2);
    Stride strideB = make_Coord(ldb0, ldb1, ldb2);
    Stride strideC = make_Coord(ldc0, ldc1, ldc2);

    // Define a CUTLASS GEMM type
    CutlassConv conv_operator;

    ///Creat the arguments struct from input
    CutlassConv::Arguments args({NX*NY, NN*NF, NC*NR*NS},  // Conv Problem dimensions
                                {},          // padding ...
                                {A, strideA},    // Tensor-ref for source matrix A
                                {B, strideB},    // Tensor-ref for source matrix B
                                {C, strideC},    // Tensor-ref for source matrix C
                                {alpha, beta}); // Scalars used in the Epilogue


    /// Launch the CUTLASS GEMM kernel.
    cutlass::Status status = conv_operator(args);

    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }

    // Return success, if no errors were encountered.
    return hipSuccess;
}

///Yufan: check it later to modify
///////////////////////////////////////////////////////////////////////////////////////////////////
//
// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Kernel to initialize a matrix with small integers.
__global__ void InitializeMatrix_kernel(
        float *matrix,
        int ldm0, int ldm1, int ldm2, int outer,
        int seed = 0) {

    int third_d = ldm2/ldm1/ldm0;
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    
    for (int fourthD = 0; fourthD < outer; ++fourthD){
        for(int thirdD = 0; thirdD < third_d; ++thirdD){
            if (i < ldm0 && j < ldm1/ldm0) {
                int offset = i+ j*ldm0 + thirdD*ldm1 + fourthD*ldm2; ///Yufan: Since 4D input 
                // Generate arbitrary elements.
                int const k = 16807;
                int const m = 16;
                float value = float(((offset + seed) * k % m) - m / 2);

                matrix[offset] = value;
            }
        }
    }
}

/// Simple function to initialize a matrix to arbitrary small integers.

hipError_t InitializeMatrix(float *matrix, int ldm0, int ldm1, int ldm2, int outer, int seed = 0) {

    dim3 block(16, 16);
    dim3 grid(
            (ldm0 + block.x - 1) / block.x,
            (ldm1/ldm0 + block.y - 1) / block.y
    );

    InitializeMatrix_kernel << < grid, block >> > (matrix, ldm0, ldm1, ldm2, outer, seed);

    return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocates device memory for a matrix then fills with arbitrary small integers.
hipError_t AllocateMatrix(float **matrix, int ldm0, int ldm1, int ldm2, int outer, int seed = 0) {
    hipError_t result;

    size_t sizeof_matrix = sizeof(float) * outer * ldm2;

    // Allocate device memory.
    result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

    if (result != hipSuccess) {
        std::cerr << "Failed to allocate matrix: "
                  << hipGetErrorString(result) << std::endl;
        return result;
    }

    // Clear the allocation.
    result = hipMemset(*matrix, 0, sizeof_matrix);

    if (result != hipSuccess) {
        std::cerr << "Failed to clear matrix device memory: "
                  << hipGetErrorString(result) << std::endl;
        return result;
    }

    // Initialize matrix elements to arbitrary small integers.
    result = InitializeMatrix(*matrix, ldm0, ldm1, ldm2, outer, seed);

    if (result != hipSuccess) {
        std::cerr << "Failed to initialize matrix: "
                  << hipGetErrorString(result) << std::endl;
        return result;
    }

    return result;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Naive reference GEMM computation.
__global__ void ReferenceConv_kernel(
        int NF, int NY, int NX, int NH, int NW, int NR, int NS, int NC,
        int sW, int sH,
        float alpha,
        float const *A,     //Input
        float const *B,     //Kernel
        float beta,
        float *C) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < M && j < N) {
        float accumulator = 0;

        int x = i % NX;
        int y = i / NX;
        int f = j % NF;
        int n = j / NF;

        for (int c = 0; c < NC; ++c) {
            for (int r = 0; r < NR; ++r) {
                for (int s = 0; s < NS; ++s) {
                    /*Output[n][k][y][x] += Input[n][c][y*StrideV+r][x*StrideH+s] * Kernel[k][c][r][s];*/
                    C[n * NF * NY * NX + f * NY * NX + y * NX + x] +=
                            A[n * NC * NH * NW + c * NH * NW + (y * sH + r) * NW + (x * sW + s)] *
                            B[f * NC * NR * NS + c * NR * NS + r * NS + s];
                }
            }
        }
    }
}

/// Reference GEMM computation.
hipError_t ReferenceConv(
        int NF, int NY, int NX, int NH, int NW, int NR, int NS, int NC,
        int sW, int sH,
        float alpha,
        float const *A,     //Input
        float const *B,     //Kernel
        float beta,
        float *C) {

    dim3 block(16, 16);
    dim3 grid(
            (M + block.x - 1) / block.x,
            (N + block.y - 1) / block.y
    );

    ReferenceConv_kernel << < grid, block >> > (NF, NY, NX, NH, NW, NR, NS,NC,
            sW, sH, alpha, A, B, beta, C);

    return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
hipError_t TestCutlassConv(int NW, int NH, int NC, int NN, int NF, int NR, int NS, /*input and kernel size*/
        int pH, int pW, int sH, int sW, int dH, int dW /*padding ...*/
        float alpha, float beta) {
    hipError_t result;

    //
    // Define several matrices to be used as operands to GEMM kernels.
    //

    // Compute leading dimensions for each matrix.


    int lda0 = NW;       //first stride (FVI) along index H
    int lda1 = NW*NH;     //second stride along index C
    int lda2 = NC*NW*NH;   //third stride along index B
    
    int ldb0 = NS;
    int ldb1 = NS*NR;
    int ldb2 = NS*NR*NC;
    
    int X = (NW+2*pW-NS)/sW+1;
    int Y = (NH+2*pH-NR)/sH+1;
    int ldc0 = NX;
    int ldc1 = NX*NY;
    int ldc2 = NX*NY*NF;

    // Compute size in bytes of the C matrix.
    size_t sizeof_C = sizeof(float) * NX*NY*NF*NN;

    // Define pointers to matrices in GPU device memory.
    float *A;
    float *B;
    float *C_cutlass;
    float *C_reference;

    //
    // Allocate matrices in GPU device memory with arbitrary seeds.
    //

    result = AllocateMatrix(&A, lda0, lda1, lda2, NN, 0);

    if (result != hipSuccess) {
        return result;
    }

    result = AllocateMatrix(&B, ldb0, ldb1, ldb2, NF, 17);

    if (result != hipSuccess) {
        hipFree(A);
        return result;
    }

    result = AllocateMatrix(&C_cutlass, ldc0, ldc1, ldc2, NN, 101);

    if (result != hipSuccess) {
        hipFree(A);
        hipFree(B);
        return result;
    }

    result = AllocateMatrix(&C_reference, ldc0, ldc1, ldc2, NN, 101);

    if (result != hipSuccess) {
        hipFree(A);
        hipFree(B);
        hipFree(C_cutlass);
        return result;
    }

    result = hipMemcpy(C_reference, C_cutlass, sizeof_C, hipMemcpyDeviceToDevice);

    if (result != hipSuccess) {
        std::cerr << "Failed to copy C_cutlass matrix to C_reference: "
                  << hipGetErrorString(result) << std::endl;

        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);

        return result;
    }

    //
    // Launch CUTLASS GEMM.
    //
    ///Yufan: need to change
    result = CutlassSconvNN(M, N, K, alpha, A, lda0, lda1, lda2, B, ldb0, ldb1, ldb2, beta, C_cutlass, ldc0, ldc1, ldc2);

    if (result != hipSuccess) {
        std::cerr << "CUTLASS GEMM kernel failed: "
                  << hipGetErrorString(result) << std::endl;

        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);

        return result;
    }

    //
    // Verify.
    //
    // Launch reference CONV
    result = ReferenceConv(NF, NY, NX, NH, NW, NR, NS,NC,
                           sW, sH, alpha, A, B, beta, C_reference);

    if (result != hipSuccess) {
        std::cerr << "Reference GEMM kernel failed: "
                  << hipGetErrorString(result) << std::endl;

        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);

        return result;
    }

    // Copy to host and verify equivalence.
    std::vector<float> host_cutlass(NX*NY*NF*NN, 0);
    std::vector<float> host_reference(NX*NY*NF*NN, 0);

    result = hipMemcpy(host_cutlass.data(), C_cutlass, sizeof_C, hipMemcpyDeviceToHost);

    if (result != hipSuccess) {
        std::cerr << "Failed to copy CUTLASS GEMM results: "
                  << hipGetErrorString(result) << std::endl;

        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);

        return result;
    }

    result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);

    if (result != hipSuccess) {
        std::cerr << "Failed to copy Reference GEMM results: "
                  << hipGetErrorString(result) << std::endl;

        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);

        return result;
    }

    //
    // Free device memory allocations.
    //

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    //
    // Test for bit equivalence of results.
    //

//  for(std::vector<float>::iterator it = host_cutlass.begin(); it != host_cutlass.end(); it++){
//      printf("value = %0.2f\n", *it);
//  }
    if (host_cutlass != host_reference) {
        std::cerr << "CUTLASS results incorrect." << std::endl;

        return hipErrorUnknown;
    }

    return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Entry point to basic_conv example.
//
// usage:
//
//   00_basic_conv <M> <N> <K> <alpha> <beta>
//
int main(int argc, const char *arg[]) {

    //
    // Parse the command line to obtain GEMM dimensions and scalar values.
    //

    // GEMM problem dimensions.
    int problem[3] = {128, 64, 32};

    for (int i = 1; i < argc && i < 4; ++i) {
        std::stringstream ss(arg[i]);
        ss >> problem[i - 1];
    }

    // Scalars used for linear scaling the result of the matrix product.
    float scalars[2] = {1, 0};

    for (int i = 4; i < argc && i < 6; ++i) {
        std::stringstream ss(arg[i]);
        ss >> scalars[i - 4];
    }

    //
    // Run the CUTLASS GEMM test.
    //

    hipError_t result = TestCutlassConv(
            problem[0],     // GEMM M dimension
            problem[1],     // GEMM N dimension
            problem[2],     // GEMM K dimension
            scalars[0],     // alpha
            scalars[1]      // beta
    );

    if (result == hipSuccess) {
        std::cout << "Passed." << std::endl;
    }

    // Exit.
    return result == hipSuccess ? 0 : -1;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
